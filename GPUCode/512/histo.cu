#include "hip/hip_runtime.h"
#include <stdio.h>
#include "histo.h"

__global__ void histo_kernel(int histo_size, double reqNumThreads, int numPixPerThread, int numElemPerThread, int imgDataBufferSize, float *dev_imgDataBuffer, int nbBinsPerDim, int binSize, int *dev_histo){
    int stride = 0;
	int const part_histo_size = 12288;
	 int stride_offset = part_histo_size;
	 int histo_count = 1;

	__shared__ int temp_histo[part_histo_size];

	while(histo_count <= (((histo_size - 1)/part_histo_size)+1)){
    	temp_histo[threadIdx.x * 24] = 0;
    	temp_histo[threadIdx.x * 24 + 1] = 0;
    	temp_histo[threadIdx.x * 24 + 2] = 0;
    	temp_histo[threadIdx.x * 24 + 3] = 0;
    	temp_histo[threadIdx.x * 24 + 4] = 0;
    	temp_histo[threadIdx.x * 24 + 5] = 0;
    	temp_histo[threadIdx.x * 24 + 6] = 0;
    	temp_histo[threadIdx.x * 24 + 7] = 0;
    	temp_histo[threadIdx.x * 24 + 8] = 0;
    	temp_histo[threadIdx.x * 24 + 9] = 0;
    	temp_histo[threadIdx.x * 24 + 10] = 0;
    	temp_histo[threadIdx.x * 24 + 11] = 0;
    	temp_histo[threadIdx.x * 24 + 12] = 0;
    	temp_histo[threadIdx.x * 24 + 13] = 0;
    	temp_histo[threadIdx.x * 24 + 14] = 0;
    	temp_histo[threadIdx.x * 24 + 15] = 0;
    	temp_histo[threadIdx.x * 24 + 16] = 0;
    	temp_histo[threadIdx.x * 24 + 17] = 0;
    	temp_histo[threadIdx.x * 24 + 18] = 0;
    	temp_histo[threadIdx.x * 24 + 19] = 0;
    	temp_histo[threadIdx.x * 24 + 20] = 0;
    	temp_histo[threadIdx.x * 24 + 21] = 0;
    	temp_histo[threadIdx.x * 24 + 22] = 0;
    	temp_histo[threadIdx.x * 24 + 23] = 0;


    	__syncthreads();

    	int i = threadIdx.x + blockIdx.x * blockDim.x;
    	int offset = blockDim.x * gridDim.x;
    	while(i < reqNumThreads){
    	//	int j = 0;
    	//	while((j < numPixPerThread) && (((i * numElemPerThread)+(j * 3)) < imgDataBufferSize)){
    			//printf("Inside while\n");
    			float L = dev_imgDataBuffer[(i * numElemPerThread)];
    			float a = dev_imgDataBuffer[(i * numElemPerThread + 1)];
    			float b = dev_imgDataBuffer[(i * numElemPerThread + 2)];
    			int idx = ((((int)round(L)-0)/binSize)*nbBinsPerDim*nbBinsPerDim)+
    					(((int)round(a)+127)/binSize)*nbBinsPerDim +
    					((int)round(b)+127)/binSize;
    			/**checking if the idx lies between the current range of histogram**/
    			if(idx >= stride && idx < stride_offset){
    				int finalIdx = idx - stride;
    				atomicAdd(&temp_histo[finalIdx],1);
    			}//end of if condition
    		//	j++;
    		//}//end of while loop calculating number of n values per thread
    		i += offset;
    	}//end of image scan while condition
    	__syncthreads();

    		atomicAdd(&dev_histo[(threadIdx.x * 24) + stride], temp_histo[threadIdx.x * 24 + 0]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 1) + stride], temp_histo[threadIdx.x * 24 + 1]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 2) + stride], temp_histo[threadIdx.x * 24 + 2]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 3) + stride], temp_histo[threadIdx.x * 24 + 3]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 4) + stride], temp_histo[threadIdx.x * 24 + 4]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 5) + stride], temp_histo[threadIdx.x * 24 + 5]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 6) + stride], temp_histo[threadIdx.x * 24 + 6]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 7) + stride], temp_histo[threadIdx.x * 24 + 7]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 8) + stride], temp_histo[threadIdx.x * 24 + 8]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 9) + stride], temp_histo[threadIdx.x * 24 + 9]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 10) + stride], temp_histo[threadIdx.x * 24 + 10]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 11) + stride], temp_histo[threadIdx.x * 24 + 11]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 12) + stride], temp_histo[threadIdx.x * 24 + 12]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 13) + stride], temp_histo[threadIdx.x * 24 + 13]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 14) + stride], temp_histo[threadIdx.x * 24 + 14]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 15) + stride], temp_histo[threadIdx.x * 24 + 15]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 16) + stride], temp_histo[threadIdx.x * 24 + 16]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 17) + stride], temp_histo[threadIdx.x * 24 + 17]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 18) + stride], temp_histo[threadIdx.x * 24 + 18]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 19) + stride], temp_histo[threadIdx.x * 24 + 19]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 20) + stride], temp_histo[threadIdx.x * 24 + 20]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 21) + stride], temp_histo[threadIdx.x * 24 + 21]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 22) + stride], temp_histo[threadIdx.x * 24 + 22]);
    		atomicAdd(&dev_histo[(threadIdx.x * 24 + 23) + stride], temp_histo[threadIdx.x * 24 + 23]);

    	stride_offset += part_histo_size;
    	stride += part_histo_size;
    	histo_count++;
	}/** end of histo_count while condition **/
}/** end of kernel function **/


void createHisto(float *imgDataBuffer, int imgDataBufferSize, int nbBinsPerDim, int imgSize, int binSize, int *count){
hipEvent_t hostDeviceStart,hostDeviceStop;
hipEventCreate( &hostDeviceStart );
hipEventCreate( &hostDeviceStop );
hipEventRecord( hostDeviceStart, 0 );
float *dev_imgDataBuffer;
hipError_t errorOne = hipMalloc((void**)&dev_imgDataBuffer, imgDataBufferSize*sizeof(float));
if(errorOne != hipSuccess)
{
  printf("CUDA errorOne: %s\n", hipGetErrorString(errorOne));
}
hipMemcpy(dev_imgDataBuffer, imgDataBuffer, imgDataBufferSize*sizeof(float), hipMemcpyHostToDevice);

int *dev_histo;
hipMalloc((void**)&dev_histo, nbBinsPerDim*nbBinsPerDim*nbBinsPerDim*sizeof(int));

hipMemset(dev_histo, 0, nbBinsPerDim*nbBinsPerDim*nbBinsPerDim*sizeof(int));
hipEventRecord( hostDeviceStop, 0 );
hipEventSynchronize( hostDeviceStop );
float hostDeviceElapsedTime;
hipEventElapsedTime( &hostDeviceElapsedTime,hostDeviceStart, hostDeviceStop );
printf( "\nTime to transfer host to device:  %3.1f ms\n", hostDeviceElapsedTime );

int numPixPerThread = 1;
int numElemPerThread = numPixPerThread*3;
int histo_size = nbBinsPerDim*nbBinsPerDim*nbBinsPerDim;
double reqNumThreads = (((imgSize - 1)/numPixPerThread)+1);
size_t printBufferSize = 1048576*100;
hipDeviceSetLimit(hipLimitPrintfFifoSize, printBufferSize);
hipEvent_t start,stop;
hipEventCreate( &start );
hipEventCreate( &stop );
hipEventRecord( start, 0 );
histo_kernel<<<26, 512>>>(histo_size, reqNumThreads, numPixPerThread, numElemPerThread, imgDataBufferSize, dev_imgDataBuffer, nbBinsPerDim, binSize, dev_histo);
hipEventRecord( stop, 0 );
hipEventSynchronize( stop );
float elapsedTime;
hipEventElapsedTime( &elapsedTime,start, stop );
printf( "Time to generate:  %3.1f ms\n", elapsedTime );
hipDeviceSynchronize();
hipError_t error = hipGetLastError();
if(error != hipSuccess)
{
  // print the CUDA error message and exit
  printf("CUDA error: %s\n", hipGetErrorString(error));
  printf("I am inside");
}
//int histo[nbBinsPerDim*nbBinsPerDim*nbBinsPerDim];
hipEvent_t deviceToHostStart,deviceToHostStop;
hipEventCreate( &deviceToHostStart );
hipEventCreate( &deviceToHostStop );
hipEventRecord( deviceToHostStart, 0 );
hipMemcpy(count, dev_histo, nbBinsPerDim * nbBinsPerDim * nbBinsPerDim*sizeof(int), hipMemcpyDeviceToHost);
hipEventRecord( deviceToHostStop, 0 );
hipEventSynchronize( deviceToHostStop );
float deviceToHostElapsedTime;
hipEventElapsedTime( &deviceToHostElapsedTime,deviceToHostStart, deviceToHostStop );
printf( "\nTime to transfer device to host:  %3.1f ms\n", deviceToHostElapsedTime );
hipFree(dev_histo);
hipFree(dev_imgDataBuffer);
//return histo;
FILE *histoNonZeroCU;
histoNonZeroCU = fopen("histoNonZeroCU.txt", "a");
for(int i = 0; i < nbBinsPerDim * nbBinsPerDim * nbBinsPerDim; i++){
	if(count[i] != 0)
	fprintf(histoNonZeroCU, "%d) %d \n",i+1, count[i]);
}
fclose(histoNonZeroCU);


}
